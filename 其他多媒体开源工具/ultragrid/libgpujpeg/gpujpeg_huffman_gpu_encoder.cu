#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2011, CESNET z.s.p.o
 * Copyright (c) 2011, Silicon Genome, LLC.
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */
 
#include "gpujpeg_huffman_gpu_encoder.h"
#include "gpujpeg_util.h"

#define THREAD_BLOCK_SIZE 48

#ifdef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
/** Allocate huffman tables in constant memory */
__constant__ struct gpujpeg_table_huffman_encoder gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_TYPE_COUNT][GPUJPEG_HUFFMAN_TYPE_COUNT];
/** Pass huffman tables to encoder */
extern struct gpujpeg_table_huffman_encoder (*gpujpeg_encoder_table_huffman)[GPUJPEG_COMPONENT_TYPE_COUNT][GPUJPEG_HUFFMAN_TYPE_COUNT] = &gpujpeg_huffman_gpu_encoder_table_huffman;
#endif

/** Natural order in constant memory */
__constant__ int gpujpeg_huffman_gpu_encoder_order_natural[GPUJPEG_ORDER_NATURAL_SIZE];

/**
 * Write one byte to compressed data
 * 
 * @param data_compressed  Data compressed
 * @param value  Byte value to write
 * @return void
 */
#define gpujpeg_huffman_gpu_encoder_emit_byte(data_compressed, value) { \
    *data_compressed = (uint8_t)(value); \
    data_compressed++; }
    
/**
 * Write two bytes to compressed data
 * 
 * @param data_compressed  Data compressed
 * @param value  Two-byte value to write
 * @return void
 */
#define gpujpeg_huffman_gpu_encoder_emit_2byte(data_compressed, value) { \
    *data_compressed = (uint8_t)(((value) >> 8) & 0xFF); \
    data_compressed++; \
    *data_compressed = (uint8_t)((value) & 0xFF); \
    data_compressed++; }
    
/**
 * Write marker to compressed data
 * 
 * @param data_compressed  Data compressed
 * @oaran marker  Marker to write (JPEG_MARKER_...)
 * @return void
 */
#define gpujpeg_huffman_gpu_encoder_marker(data_compressed, marker) { \
    *data_compressed = 0xFF;\
    data_compressed++; \
    *data_compressed = (uint8_t)(marker); \
    data_compressed++; }

/**
 * Output bits to the file. Only the right 24 bits of put_buffer are used; 
 * the valid bits are left-justified in this part.  At most 16 bits can be 
 * passed to EmitBits in one call, and we never retain more than 7 bits 
 * in put_buffer between calls, so 24 bits are sufficient.
 * 
 * @param coder  Huffman coder structure
 * @param code  Huffman code
 * @param size  Size in bits of the Huffman code
 * @return void
 */
__device__ inline int
gpujpeg_huffman_gpu_encoder_emit_bits(unsigned int code, int size, int & put_value, int & put_bits, uint8_t* & data_compressed)
{
    // This routine is heavily used, so it's worth coding tightly
    int _put_buffer = (int)code;
    int _put_bits = put_bits;
    // If size is 0, caller used an invalid Huffman table entry
    if ( size == 0 )
        return -1;
    // Mask off any extra bits in code
    _put_buffer &= (((int)1) << size) - 1; 
    // New number of bits in buffer
    _put_bits += size;                    
    // Align incoming bits
    _put_buffer <<= 24 - _put_bits;        
    // And merge with old buffer contents
    _put_buffer |= put_value;    
    // If there are more than 8 bits, write it out
    unsigned char uc;
    while ( _put_bits >= 8 ) {
        // Write one byte out
        uc = (unsigned char) ((_put_buffer >> 16) & 0xFF);
        gpujpeg_huffman_gpu_encoder_emit_byte(data_compressed, uc);
        // If need to stuff a zero byte
        if ( uc == 0xFF ) {  
            // Write zero byte out
            gpujpeg_huffman_gpu_encoder_emit_byte(data_compressed, 0);
        }
        _put_buffer <<= 8;
        _put_bits -= 8;
    }
    // update state variables
    put_value = _put_buffer; 
    put_bits = _put_bits;
    return 0;
}

/**
 * Emit left bits
 * 
 * @param coder  Huffman coder structure
 * @return void
 */
__device__ inline void
gpujpeg_huffman_gpu_encoder_emit_left_bits(int & put_value, int & put_bits, uint8_t* & data_compressed)
{
    // Fill 7 bits with ones
    if ( gpujpeg_huffman_gpu_encoder_emit_bits(0x7F, 7, put_value, put_bits, data_compressed) != 0 )
        return;
    
    //unsigned char uc = (unsigned char) ((put_value >> 16) & 0xFF);
    // Write one byte out
    //gpujpeg_huffman_gpu_encoder_emit_byte(data_compressed, uc);
    
    put_value = 0; 
    put_bits = 0;
}

/**
 * Encode one 8x8 block
 *
 * @return 0 if succeeds, otherwise nonzero
 */
__device__ int
gpujpeg_huffman_gpu_encoder_encode_block(int & put_value, int & put_bits, int & dc, int16_t* data, uint8_t* & data_compressed, 
    struct gpujpeg_table_huffman_encoder* d_table_dc, struct gpujpeg_table_huffman_encoder* d_table_ac)
{
    typedef uint64_t loading_t;
    const int loading_iteration_count = 64 * 2 / sizeof(loading_t);
    
    // Load block to shared memory
    __shared__ int16_t s_data[64 * THREAD_BLOCK_SIZE];
    for ( int i = 0; i < loading_iteration_count; i++ ) {
        ((loading_t*)s_data)[loading_iteration_count * threadIdx.x + i] = ((loading_t*)data)[i];
    }
    int data_start = 64 * threadIdx.x;

    // Encode the DC coefficient difference per section F.1.2.1
    int temp = s_data[data_start + 0] - dc;
    dc = s_data[data_start + 0];
    
    int temp2 = temp;
    if ( temp < 0 ) {
        // Temp is abs value of input
        temp = -temp;
        // For a negative input, want temp2 = bitwise complement of abs(input)
        // This code assumes we are on a two's complement machine
        temp2--;
    }

    // Find the number of bits needed for the magnitude of the coefficient
    int nbits = 0;
    while ( temp ) {
        nbits++;
        temp >>= 1;
    }

    // Write category number
    if ( gpujpeg_huffman_gpu_encoder_emit_bits(d_table_dc->code[nbits], d_table_dc->size[nbits], put_value, put_bits, data_compressed) != 0 ) {
        return -1;
    }

    // Write category offset (EmitBits rejects calls with size 0)
    if ( nbits ) {
        if ( gpujpeg_huffman_gpu_encoder_emit_bits((unsigned int) temp2, nbits, put_value, put_bits, data_compressed) != 0 )
            return -1;
    }
    
    // Encode the AC coefficients per section F.1.2.2 (r = run length of zeros)
    int r = 0;
    for ( int k = 1; k < 64; k++ ) 
    {
        temp = s_data[data_start + gpujpeg_huffman_gpu_encoder_order_natural[k]];
        if ( temp == 0 ) {
            r++;
        }
        else {
            // If run length > 15, must emit special run-length-16 codes (0xF0)
            while ( r > 15 ) {
                if ( gpujpeg_huffman_gpu_encoder_emit_bits(d_table_ac->code[0xF0], d_table_ac->size[0xF0], put_value, put_bits, data_compressed) != 0 )
                    return -1;
                r -= 16;
            }

            temp2 = temp;
            if ( temp < 0 ) {
                // temp is abs value of input
                temp = -temp;        
                // This code assumes we are on a two's complement machine
                temp2--;
            }

            // Find the number of bits needed for the magnitude of the coefficient
            // there must be at least one 1 bit
            nbits = 1;
            while ( (temp >>= 1) )
                nbits++;

            // Emit Huffman symbol for run length / number of bits
            int i = (r << 4) + nbits;
            if ( gpujpeg_huffman_gpu_encoder_emit_bits(d_table_ac->code[i], d_table_ac->size[i], put_value, put_bits, data_compressed) != 0 )
                return -1;

            // Write Category offset
            if ( gpujpeg_huffman_gpu_encoder_emit_bits((unsigned int) temp2, nbits, put_value, put_bits, data_compressed) != 0 )
                return -1;

            r = 0;
        }
    }

    // If all the left coefs were zero, emit an end-of-block code
    if ( r > 0 ) {
        if ( gpujpeg_huffman_gpu_encoder_emit_bits(d_table_ac->code[0], d_table_ac->size[0], put_value, put_bits, data_compressed) != 0 )
            return -1;
    }

    return 0;
}

/**
 * Huffman encoder kernel
 * 
 * @return void
 */
__global__ void
gpujpeg_huffman_encoder_encode_kernel(
    struct gpujpeg_component* d_component,
    struct gpujpeg_segment* d_segment,
    int comp_count,
    int segment_count, 
    uint8_t* d_data_compressed
#ifndef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
    ,struct gpujpeg_table_huffman_encoder* d_table_y_dc
    ,struct gpujpeg_table_huffman_encoder* d_table_y_ac
    ,struct gpujpeg_table_huffman_encoder* d_table_cbcr_dc
    ,struct gpujpeg_table_huffman_encoder* d_table_cbcr_ac
#endif
)
{    
#ifdef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
    // Get huffman tables from constant memory
    struct gpujpeg_table_huffman_encoder* d_table_y_dc = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_DC];
    struct gpujpeg_table_huffman_encoder* d_table_y_ac = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_AC];
    struct gpujpeg_table_huffman_encoder* d_table_cbcr_dc = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_DC];
    struct gpujpeg_table_huffman_encoder* d_table_cbcr_ac = &gpujpeg_huffman_gpu_encoder_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_AC];
#endif
    
    int segment_index = blockIdx.x * blockDim.x + threadIdx.x;
    if ( segment_index >= segment_count )
        return;
    
    struct gpujpeg_segment* segment = &d_segment[segment_index];
    
    // Initialize huffman coder
    int put_value = 0;
    int put_bits = 0;
    int dc[GPUJPEG_MAX_COMPONENT_COUNT];
    for ( int comp = 0; comp < GPUJPEG_MAX_COMPONENT_COUNT; comp++ )
        dc[comp] = 0;
    
    // Prepare data pointers
    uint8_t* data_compressed = &d_data_compressed[segment->data_compressed_index];
    uint8_t* data_compressed_start = data_compressed;
    
    // Non-interleaving mode
    if ( comp_count == 1 ) {
        int segment_index = segment->scan_segment_index;
        // Encode MCUs in segment
        for ( int mcu_index = 0; mcu_index < segment->mcu_count; mcu_index++ ) {
            // Get component for current scan
            struct gpujpeg_component* component = &d_component[segment->scan_index];
     
            // Get component data for MCU
            int16_t* block = &component->d_data_quantized[(segment_index * component->segment_mcu_count + mcu_index) * component->mcu_size];
            
            // Get coder parameters
            int & component_dc = dc[segment->scan_index];
            
            // Get huffman tables
            struct gpujpeg_table_huffman_encoder* d_table_dc = NULL;
            struct gpujpeg_table_huffman_encoder* d_table_ac = NULL;
            if ( component->type == GPUJPEG_COMPONENT_LUMINANCE ) {
                d_table_dc = d_table_y_dc;
                d_table_ac = d_table_y_ac;
            } else {
                d_table_dc = d_table_cbcr_dc;
                d_table_ac = d_table_cbcr_ac;
            }
            
            // Encode 8x8 block
            if ( gpujpeg_huffman_gpu_encoder_encode_block(put_value, put_bits, component_dc, block, data_compressed, d_table_dc, d_table_ac) != 0 )
                break;
        } 
    }
    // Interleaving mode
    else {
        int segment_index = segment->scan_segment_index;
        // Encode MCUs in segment
        for ( int mcu_index = 0; mcu_index < segment->mcu_count; mcu_index++ ) {
            //assert(segment->scan_index == 0);
            for ( int comp = 0; comp < comp_count; comp++ ) {
                struct gpujpeg_component* component = &d_component[comp];

                // Prepare mcu indexes
                int mcu_index_x = (segment_index * component->segment_mcu_count + mcu_index) % component->mcu_count_x;
                int mcu_index_y = (segment_index * component->segment_mcu_count + mcu_index) / component->mcu_count_x;
                // Compute base data index
                int data_index_base = mcu_index_y * (component->mcu_size * component->mcu_count_x) + mcu_index_x * (component->mcu_size_x * GPUJPEG_BLOCK_SIZE);
                
                // For all vertical 8x8 blocks
                for ( int y = 0; y < component->sampling_factor.vertical; y++ ) {
                    // Compute base row data index
                    int data_index_row = data_index_base + y * (component->mcu_count_x * component->mcu_size_x * GPUJPEG_BLOCK_SIZE);
                    // For all horizontal 8x8 blocks
                    for ( int x = 0; x < component->sampling_factor.horizontal; x++ ) {
                        // Compute 8x8 block data index
                        int data_index = data_index_row + x * GPUJPEG_BLOCK_SIZE * GPUJPEG_BLOCK_SIZE;
                        
                        // Get component data for MCU
                        int16_t* block = &component->d_data_quantized[data_index];
                        
                        // Get coder parameters
                        int & component_dc = dc[comp];
            
                        // Get huffman tables
                        struct gpujpeg_table_huffman_encoder* d_table_dc = NULL;
                        struct gpujpeg_table_huffman_encoder* d_table_ac = NULL;
                        if ( component->type == GPUJPEG_COMPONENT_LUMINANCE ) {
                            d_table_dc = d_table_y_dc;
                            d_table_ac = d_table_y_ac;
                        } else {
                            d_table_dc = d_table_cbcr_dc;
                            d_table_ac = d_table_cbcr_ac;
                        }
                        
                        // Encode 8x8 block
                        gpujpeg_huffman_gpu_encoder_encode_block(put_value, put_bits, component_dc, block, data_compressed, d_table_dc, d_table_ac);
                    }
                }
            }
        }
    }
    
    // Emit left bits
    if ( put_bits > 0 )
        gpujpeg_huffman_gpu_encoder_emit_left_bits(put_value, put_bits, data_compressed);

    // Output restart marker
    int restart_marker = GPUJPEG_MARKER_RST0 + (segment->scan_segment_index % 8);
    gpujpeg_huffman_gpu_encoder_marker(data_compressed, restart_marker);
                
    // Set compressed size
    segment->data_compressed_size = data_compressed - data_compressed_start;
}

/** Documented at declaration */
int
gpujpeg_huffman_gpu_encoder_init()
{
    // Copy natural order to constant device memory
    hipMemcpyToSymbol(
        (const char*)gpujpeg_huffman_gpu_encoder_order_natural,
        gpujpeg_order_natural, 
        GPUJPEG_ORDER_NATURAL_SIZE * sizeof(int),
        0,
        hipMemcpyHostToDevice
    );
    gpujpeg_cuda_check_error("Huffman encoder init");
    
    return 0;
}

/** Documented at declaration */
int
gpujpeg_huffman_gpu_encoder_encode(struct gpujpeg_encoder* encoder)
{    
    // Get coder
    struct gpujpeg_coder* coder = &encoder->coder;
    
    assert(coder->param.restart_interval > 0);
    
    int comp_count = 1;
    if ( coder->param.interleaved == 1 )
        comp_count = coder->param_image.comp_count;
    assert(comp_count >= 1 && comp_count <= GPUJPEG_MAX_COMPONENT_COUNT);
            
    // Run kernel
    dim3 thread(THREAD_BLOCK_SIZE);
    dim3 grid(gpujpeg_div_and_round_up(coder->segment_count, thread.x));
    gpujpeg_huffman_encoder_encode_kernel<<<grid, thread>>>(
        coder->d_component, 
        coder->d_segment, 
        comp_count,
        coder->segment_count, 
        coder->d_data_compressed
    #ifndef GPUJPEG_HUFFMAN_CODER_TABLES_IN_CONSTANT
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_DC]
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_AC]
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_DC]
        ,encoder->d_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_AC]
    #endif
    );
    hipDeviceSynchronize();
    gpujpeg_cuda_check_error("Huffman encoding failed");
    
    return 0;
}
